#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA_SM80
#include <nnc/gpu/3rdparty/flash_attn/flash_api.h>

static int _ccv_nnc_scaled_dot_product_attention_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	// NNC notation:
	// C = sm(Q * K^T) * V
	//
	// MFA notation:
	// O = sm(Q * K^T) * V
	assert(input_size >= 3);
	assert(output_size >= 1);
	ccv_nnc_tensor_view_t* const q = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_t* const k = (ccv_nnc_tensor_view_t*)inputs[1];
	ccv_nnc_tensor_view_t* const v = (ccv_nnc_tensor_view_t*)inputs[2];
	ccv_nnc_tensor_view_t* const attn_mask = input_size > 3 ? (ccv_nnc_tensor_view_t*)inputs[3] : 0;
	ccv_nnc_tensor_view_t* const weights = input_size > 4 ? (ccv_nnc_tensor_view_t*)inputs[4] : 0;
	ccv_nnc_tensor_view_t* const bias = input_size > 5 ? (ccv_nnc_tensor_view_t*)inputs[5] : 0;
	if (bias) // bias always requires a weight matrix.
		{ assert(weights); }

	ccv_nnc_tensor_view_t* const saved_softmax_lse = output_size > 1 ? (ccv_nnc_tensor_view_t*)outputs[1] : 0;
	ccv_nnc_tensor_view_t* const o = (weights) ? (ccv_nnc_tensor_view_t*)outputs[2] : (ccv_nnc_tensor_view_t*)outputs[0];
	const int q_nd = ccv_nnc_tensor_nd(q->info.dim);
	assert(q_nd == 3 || q_nd == 4);
	const int k_nd = ccv_nnc_tensor_nd(k->info.dim);
	assert(k_nd == 3 || k_nd == 4);
	const int v_nd = ccv_nnc_tensor_nd(v->info.dim);
	assert(v_nd == 3 || v_nd == 4);
	const int o_nd = ccv_nnc_tensor_nd(o->info.dim);
	assert(o_nd == 3 || o_nd == 4);
	assert(q_nd == k_nd && k_nd == v_nd && v_nd == o_nd);

	int qdim[CCV_NNC_MAX_DIM_ALLOC];
	int kdim[CCV_NNC_MAX_DIM_ALLOC];
	int vdim[CCV_NNC_MAX_DIM_ALLOC];
	int odim[CCV_NNC_MAX_DIM_ALLOC];
	int amdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(q, qdim);
	ccv_nnc_tensor_view_get_dim(k, kdim);
	ccv_nnc_tensor_view_get_dim(v, vdim);
	ccv_nnc_tensor_view_get_dim(o, odim);

	assert(q->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(k->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(v->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(o->info.format == CCV_TENSOR_FORMAT_NHWC);
	if (attn_mask) {
		// MFA does not support fused transposes on the mask.
		assert(attn_mask->info.format == CCV_TENSOR_FORMAT_NHWC);
	}

	assert(CCV_IS_TENSOR_CONTIGUOUS(q));
	assert(CCV_IS_TENSOR_CONTIGUOUS(k));
	assert(CCV_IS_TENSOR_CONTIGUOUS(v));
	assert(CCV_IS_TENSOR_CONTIGUOUS(o));

	if (attn_mask) {
		assert(CCV_IS_TENSOR_CONTIGUOUS(attn_mask));
	}

	int batch_size;
	int R;
	int C;
	int Hq;
	int Hk;
	int D;
	if (q_nd == 3) {
		batch_size = qdim[1];
		assert(batch_size == kdim[1]);
		R = qdim[2];
		C = kdim[2];
		Hq = Hk = 1;
		D = qdim[3];
		assert(D == kdim[3]);
	} else if (q_nd == 4) {
		batch_size = qdim[0];
		assert(batch_size == kdim[0]);
		R = qdim[1];
		C = kdim[1];
		Hq = qdim[2];
		Hk = kdim[2];
		assert(Hq >= Hk);
		assert(Hq % Hk == 0);
		D = qdim[3];
		assert(D == kdim[3]);
	}

	if (attn_mask) {
		// MFA can support am_nd == 2 and broadcast batch=1 -> batch=batch_size, but
		// wait until that occurs in practice before doing so.
		const int am_nd = ccv_nnc_tensor_nd(attn_mask->info.dim);
		assert(am_nd == 3 || am_nd == 4); // [batch_size, R, C]

		// MFA does not support attention mask broadcasting (where the R dimension
		// of Q > 1, but the R dimension of the mask == 1).
		ccv_nnc_tensor_view_get_dim(attn_mask, amdim);
		if (am_nd == 3)
		{
			assert(amdim[1] == batch_size || amdim[1] == 1);
			amdim[0] = amdim[1];
			amdim[1] = 1;
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		} else {
			assert(amdim[0] == batch_size || amdim[0] == 1);
			assert(amdim[1] == 1);
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		}
	}
	int weights_datatype = 0;
	if (weights)
		weights_datatype = CCV_GET_DATA_TYPE(weights->info.datatype) == CCV_QX ? ((weights->info.datatype & 0xff) << 12) : weights->info.datatype;

	const int is_same_dtype =
		(q->info.datatype == k->info.datatype) &&
		(q->info.datatype == v->info.datatype) &&
		(q->info.datatype == o->info.datatype) &&
		(weights ? (q->info.datatype == weights_datatype) : 1) &&
		(bias ? (q->info.datatype == bias->info.datatype) : 1);

	assert(is_same_dtype);

	Flash_fwd_params params;
	memset(&params, 0, sizeof(params));
	params.is_bf16 = false;
	params.q_ptr = q->data.u8;
	params.k_ptr = k->data.u8;
	params.v_ptr = v->data.u8;
	params.q_row_stride = D * Hq;
	params.k_row_stride = D * Hk;
	params.v_row_stride = D * Hk;
	params.q_head_stride = D;
	params.k_head_stride = D;
	params.v_head_stride = D;
	params.q_batch_stride = R * Hq * D;
	params.k_batch_stride = C * Hk * D;
	params.v_batch_stride = C * Hk * D;
	auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
	params.seqlen_q = R;
	params.seqlen_q_rounded = round_multiple(R, 128);
	params.seqlen_k = C;
	params.seqlen_k_rounded = round_multiple(C, 128);
	params.d = D;
	assert(D % 8 == 0);
	params.d_rounded = round_multiple(D, 32);
	params.o_ptr = o->data.u8;
	params.o_row_stride = D * Hq;
	params.o_head_stride = D;
	params.o_batch_stride = R * Hq * D;
	params.b = batch_size;
	params.h = Hq;
	params.h_k = Hk;
	params.h_h_k_ratio = Hq / Hk;
	params.scale_softmax = cmd.info.scaled_dot_product_attention.scale;
	params.scale_softmax_log2 = cmd.info.scaled_dot_product_attention.scale * M_LOG2E;
	params.is_causal = cmd.info.scaled_dot_product_attention.is_causal;
	params.p_dropout = 1;
	params.p_dropout_in_uint8_t = 255;
	params.rp_dropout = 1;
	params.scale_softmax_rp_dropout = params.scale_softmax;
	params.window_size_left = ccv_max(R, C);
	params.window_size_right = params.is_causal ? 0 : ccv_max(R, C);
	params.is_seqlens_k_cumulative = true;
	const int block_n = D <= 64 ? 256 : (D <= 128 ? 128 : 64);
	const int num_n_blocks = (C + block_n - 1) / block_n;
	// Technically kBlockM = 64 only for the splitKV kernels, not the standard kernel.
	// In any case we don't expect seqlen_q to be larger than 64 for inference.
	const int num_m_blocks = (R + 64 - 1) / 64;
	const ccv_nnc_cuda_device_prop_t props = ccv_nnc_gpu_device_props();
	params.num_splits = num_splits_heuristic(batch_size * Hq * num_m_blocks, props.multi_processor_count, num_n_blocks, 128);
	if (saved_softmax_lse)
		params.softmax_lse_ptr = saved_softmax_lse->data.u8;
	if (params.num_splits > 1)
	{
		if (saved_softmax_lse)
		{
			float* const workspace = (float*)ccv_nnc_stream_context_get_workspace(stream_context, (params.num_splits * batch_size * Hq * R + params.num_splits * batch_size * Hq * R * params.d_rounded) * sizeof(float), CCV_TENSOR_GPU_MEMORY);
			params.softmax_lseaccum_ptr = workspace;
			params.oaccum_ptr = workspace + params.num_splits * batch_size * Hq * R;
		} else {
			float* const workspace = (float*)ccv_nnc_stream_context_get_workspace(stream_context, (batch_size * Hq * R + params.num_splits * batch_size * Hq * R + params.num_splits * batch_size * Hq * R * params.d_rounded) * sizeof(float), CCV_TENSOR_GPU_MEMORY);
			params.softmax_lse_ptr = workspace;
			params.softmax_lseaccum_ptr = workspace + batch_size * Hq * R;
			params.oaccum_ptr = workspace + batch_size * Hq * R + params.num_splits * batch_size * Hq * R;
		}
	} else if (!saved_softmax_lse) {
		void* const workspace = ccv_nnc_stream_context_get_workspace(stream_context, batch_size * Hq * R * sizeof(float), CCV_TENSOR_GPU_MEMORY);
		params.softmax_lse_ptr = workspace;
	}
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	run_mha_fwd(params, stream, false);
	CUDA_ENFORCE(hipGetLastError());
	if (weights)
	{
		const ccv_nnc_tensor_view_t* a = o;
		const ccv_nnc_tensor_view_t* w = weights;
		ccv_nnc_tensor_view_t* b = (ccv_nnc_tensor_view_t*)outputs[0];
		assert(!bias || (bias->info.dim[1] == 0 || bias->info.dim[2] == 0 || bias->info.dim[3] == 0)); // It is a 1-d array
		assert(CCV_IS_TENSOR_CONTIGUOUS(b));
		const int b_nd = ccv_nnc_tensor_nd(b->info.dim);
		assert(b_nd == 3);
		int w_batch_size, w_rows, w_cols, w_batch_inc, w_rows_inc, w_cols_inc;
		const int w_nd = ccv_nnc_tensor_nd(w->info.dim);
		const int transpose_w[2] = {
			w_nd - 2, w_nd - 1
		};
		ccv_nnc_tensor_get_matrix_params(w->info, CCV_IS_TENSOR_VIEW(w) ? w->stride : 0, w->info.dim, transpose_w, &w_batch_size, &w_rows, &w_cols, &w_batch_inc, &w_rows_inc, &w_cols_inc);
		int a_rows, a_cols;
		if (o_nd == 3) {
			a_rows = odim[1] * odim[2];
			a_cols = odim[3];
		} else if (q_nd == 4) {
			a_rows = odim[0] * odim[1];
			a_cols = odim[2] * odim[3];
		}
		int b_rows, b_cols, b_rows_inc;
		b_rows = b->info.dim[0] * b->info.dim[1];
		b_cols = b->info.dim[2];
		b_rows_inc = b_cols;
		assert(a_rows == b_rows);
		assert(a_cols == w_rows);
		assert(w_cols == b_cols);

		const hipblasOperation_t transa = HIPBLAS_OP_T;
		const hipblasOperation_t transb = HIPBLAS_OP_N;
		const int lda_inc = w_cols_inc;
		const int ldb_inc = a_cols;
		size_t w_data_size = 0;
		int w_datatype = w->info.datatype;
		if (CCV_GET_DATA_TYPE(w->info.datatype) == CCV_QX)
		{
			ccv_nnc_tensor_param_t w_params = w->info;
			w_datatype = (w_params.datatype & 0xff) << 12;
			ccv_nnc_tensor_param_t depalettize_w_params = w_params;
			depalettize_w_params.datatype = w_datatype;
			depalettize_w_params.reserved = 0;
			w_data_size = ccv_nnc_tensor_data_size(depalettize_w_params);
		}
		const size_t cublas_size = ccv_nnc_cublas_workspace_size_in_bytes(inputs, input_size, outputs, output_size);
		void* workspace = 0;
		if (w_data_size > 0)
			workspace = ccv_nnc_stream_context_get_workspace(stream_context, cublas_size + w_data_size, CCV_TENSOR_GPU_MEMORY);
		unsigned char* w_data = w->data.u8;
		if (CCV_GET_DATA_TYPE(w->info.datatype) == CCV_QX)
		{
			ccv_nnc_tensor_param_t w_params = w->info;
			const size_t count = ccv_nnc_tensor_count(w_params);
			const int qbits = (w_params.datatype & 0xf00) >> 8;
			const int number_in_blocks = w_params.reserved;
			w_data = (unsigned char*)workspace + cublas_size;
			ccv_nnc_compat_depalettize(w->data.u8, w_datatype, ccv_nnc_tensor_data_size_without_padding(w_params), qbits, number_in_blocks, w_data, count, stream_context);
		}
		hipblasHandle_t cublas = ccv_nnc_stream_context_get_cublas(stream_context);
		static const half one_f16 = 1;
		static const float one_f32 = 1;
		static const double one_f64 = 1;
		static const double zero_f64 = 0;
		const void* zero = &zero_f64;
		const void* one;
		switch (ccv_nnc_cuda_compute_datatype(b->info.datatype))
		{
			case HIPBLAS_COMPUTE_16F:
				one = &one_f16;
				break;
			case HIPBLAS_COMPUTE_32F:
			case HIPBLAS_COMPUTE_32F_FAST_TF32:
				one = &one_f32;
				break;
			case HIPBLAS_COMPUTE_64F:
				one = &one_f64;
				break;
			default:
				assert(0);
		}
		ccv_nnc_stream_context_set_cublas_workspace(cublas, stream_context, cublas_size);
		if (bias)
		{
			int bias_batch_size, bias_rows, bias_cols, bias_batch_inc, bias_rows_inc, bias_cols_inc;
			const static int no_transpose[2] = {};
			ccv_nnc_tensor_get_matrix_params(bias->info, CCV_IS_TENSOR_VIEW(bias) ? bias->stride : 0, bias->info.dim, no_transpose, &bias_batch_size, &bias_rows, &bias_cols, &bias_batch_inc, &bias_rows_inc, &bias_cols_inc);
			assert(bias_batch_size == 1);
			assert(bias_cols == b_cols);
			assert(CCV_IS_TENSOR_CONTIGUOUS(bias));
			const void* const device_ones = ccv_nnc_stream_context_get_ones(stream_context, b_rows, b->info.datatype);
			CUBLAS_ENFORCE(hipblasGemmEx(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, b_cols, b_rows, 1, one, bias->data.u8, ccv_nnc_cuda_datatype(bias->info.datatype), bias_rows_inc, device_ones, ccv_nnc_cuda_datatype(b->info.datatype), 1, zero, b->data.u8, ccv_nnc_cuda_datatype(b->info.datatype), b_rows_inc, ccv_nnc_cuda_compute_datatype(b->info.datatype), CUBLAS_GEMM_DEFAULT_TENSOR_OP));
			CUBLAS_ENFORCE(hipblasGemmEx(cublas, transa, transb, b_cols, b_rows, a_cols, one, w_data, ccv_nnc_cuda_datatype(w_datatype), lda_inc, a->data.u8, ccv_nnc_cuda_datatype(a->info.datatype), ldb_inc, one, b->data.u8, ccv_nnc_cuda_datatype(b->info.datatype), b_rows_inc, ccv_nnc_cuda_compute_datatype(b->info.datatype), CUBLAS_GEMM_DEFAULT_TENSOR_OP));
		} else {
			CUBLAS_ENFORCE(hipblasGemmEx(cublas, transa, transb, b_cols, b_rows, a_cols, one, w_data, ccv_nnc_cuda_datatype(w_datatype), lda_inc, a->data.u8, ccv_nnc_cuda_datatype(a->info.datatype), ldb_inc, zero, b->data.u8, ccv_nnc_cuda_datatype(b->info.datatype), b_rows_inc, ccv_nnc_cuda_compute_datatype(b->info.datatype), CUBLAS_GEMM_DEFAULT_TENSOR_OP));
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM>
__global__ void _ccv_nnc_sum_out(const int B, const int Hk, const int r, const int D, const NUM* const a, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, B * Hk * D) {
		const int j = i / D;
		const int k = i % D;
		const NUM* const arow = a + j * r * D + k;
		float accum = (float)arow[0];
		for (int l = 1; l < r; l++)
			accum += (float)arow[l * D];
		b[i] = (NUM)accum;
	}
}

static int _ccv_nnc_scaled_dot_product_attention_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	// NNC notation:
	// C = sm(Q * K^T) * V
	//
	// MFA notation:
	// O = sm(Q * K^T) * V
	assert(input_size >= 6);
	assert(output_size >= 3);
	ccv_nnc_tensor_view_t* const d_o = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_t* const q = (ccv_nnc_tensor_view_t*)inputs[3];
	ccv_nnc_tensor_view_t* const k = (ccv_nnc_tensor_view_t*)inputs[4];
	ccv_nnc_tensor_view_t* const v = (ccv_nnc_tensor_view_t*)inputs[5];
	ccv_nnc_tensor_view_t* const attn_mask = input_size > 6 ? (ccv_nnc_tensor_view_t*)inputs[6] : 0;
	ccv_nnc_tensor_view_t* const weights = input_size > 7 ? (ccv_nnc_tensor_view_t*)inputs[7] : 0;
	ccv_nnc_tensor_view_t* const bias = input_size > 8 ? (ccv_nnc_tensor_view_t*)inputs[8] : 0;
	if (bias) // bias always requires a weight matrix.
		{ assert(weights); }
	ccv_nnc_tensor_view_t* const o = input_size > 9 ? (ccv_nnc_tensor_view_t*)inputs[9] : 0;
	ccv_nnc_tensor_view_t* const saved_softmax_lse = input_size > 10 ? (ccv_nnc_tensor_view_t*)inputs[10] : 0;
	// ccv_nnc_tensor_view_t* const qkv = input_size > 11 ? (ccv_nnc_tensor_view_t*)inputs[11] : 0;
	ccv_nnc_tensor_view_t* const dq = (ccv_nnc_tensor_view_t*)outputs[0];
	ccv_nnc_tensor_view_t* const dk = (ccv_nnc_tensor_view_t*)outputs[1];
	ccv_nnc_tensor_view_t* const dv = (ccv_nnc_tensor_view_t*)outputs[2];

	// Things we don't support.
	if (weights != 0)
		return CCV_NNC_EXEC_INVALID;

	const int q_nd = ccv_nnc_tensor_nd(q->info.dim);
	assert(q_nd == 3 || q_nd == 4);
	const int k_nd = ccv_nnc_tensor_nd(k->info.dim);
	assert(k_nd == 3 || k_nd == 4);
	const int v_nd = ccv_nnc_tensor_nd(v->info.dim);
	assert(v_nd == 3 || v_nd == 4);
	const int o_nd = ccv_nnc_tensor_nd(o->info.dim);
	assert(o_nd == 3 || o_nd == 4);
	assert(q_nd == k_nd && k_nd == v_nd && v_nd == o_nd);

	int qdim[CCV_NNC_MAX_DIM_ALLOC];
	int kdim[CCV_NNC_MAX_DIM_ALLOC];
	int vdim[CCV_NNC_MAX_DIM_ALLOC];
	int odim[CCV_NNC_MAX_DIM_ALLOC];
	int amdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(q, qdim);
	ccv_nnc_tensor_view_get_dim(k, kdim);
	ccv_nnc_tensor_view_get_dim(v, vdim);
	ccv_nnc_tensor_view_get_dim(o, odim);

	assert(q->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(k->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(v->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(o->info.format == CCV_TENSOR_FORMAT_NHWC);
	if (attn_mask) {
		// MFA does not support fused transposes on the mask.
		assert(attn_mask->info.format == CCV_TENSOR_FORMAT_NHWC);
	}

	assert(CCV_IS_TENSOR_CONTIGUOUS(q));
	assert(CCV_IS_TENSOR_CONTIGUOUS(k));
	assert(CCV_IS_TENSOR_CONTIGUOUS(v));
	assert(CCV_IS_TENSOR_CONTIGUOUS(o));
	assert(CCV_IS_TENSOR_CONTIGUOUS(d_o));
	assert(CCV_IS_TENSOR_CONTIGUOUS(saved_softmax_lse));

	if (attn_mask) {
		assert(CCV_IS_TENSOR_CONTIGUOUS(attn_mask));
	}

	int batch_size;
	int R;
	int C;
	int Hq;
	int Hk;
	int D;
	if (q_nd == 3) {
		batch_size = qdim[1];
		assert(batch_size == kdim[1]);
		R = qdim[2];
		C = kdim[2];
		Hq = Hk = 1;
		D = qdim[3];
		assert(D == kdim[3]);
	} else if (q_nd == 4) {
		batch_size = qdim[0];
		assert(batch_size == kdim[0]);
		R = qdim[1];
		C = kdim[1];
		Hq = qdim[2];
		Hk = kdim[2];
		assert(Hq >= Hk);
		assert(Hq % Hk == 0);
		D = qdim[3];
		assert(D == kdim[3]);
	}

	if (attn_mask) {
		// MFA can support am_nd == 2 and broadcast batch=1 -> batch=batch_size, but
		// wait until that occurs in practice before doing so.
		const int am_nd = ccv_nnc_tensor_nd(attn_mask->info.dim);
		assert(am_nd == 3 || am_nd == 4); // [batch_size, R, C]

		// MFA does not support attention mask broadcasting (where the R dimension
		// of Q > 1, but the R dimension of the mask == 1).
		ccv_nnc_tensor_view_get_dim(attn_mask, amdim);
		if (am_nd == 3)
		{
			assert(amdim[1] == batch_size || amdim[1] == 1);
			amdim[0] = amdim[1];
			amdim[1] = 1;
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		} else {
			assert(amdim[0] == batch_size || amdim[0] == 1);
			assert(amdim[1] == 1);
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		}
	}
	int weights_datatype = 0;
	if (weights)
		weights_datatype = CCV_GET_DATA_TYPE(weights->info.datatype) == CCV_QX ? ((weights->info.datatype & 0xff) << 12) : weights->info.datatype;

	const int is_same_dtype =
		(q->info.datatype == k->info.datatype) &&
		(q->info.datatype == v->info.datatype) &&
		(q->info.datatype == o->info.datatype) &&
		(weights ? (q->info.datatype == weights_datatype) : 1) &&
		(bias ? (q->info.datatype == bias->info.datatype) : 1);

	assert(is_same_dtype);

	Flash_bwd_params params;
	memset(&params, 0, sizeof(params));
	params.is_bf16 = false;
	params.q_ptr = q->data.u8;
	params.k_ptr = k->data.u8;
	params.v_ptr = v->data.u8;
	params.q_row_stride = D * Hq;
	params.k_row_stride = D * Hk;
	params.v_row_stride = D * Hk;
	params.q_head_stride = D;
	params.k_head_stride = D;
	params.v_head_stride = D;
	params.q_batch_stride = R * Hq * D;
	params.k_batch_stride = C * Hk * D;
	params.v_batch_stride = C * Hk * D;
	auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
	params.seqlen_q = R;
	params.seqlen_q_rounded = round_multiple(R, 128);
	params.seqlen_k = C;
	params.seqlen_k_rounded = round_multiple(C, 128);
	params.d = D;
	assert(D % 8 == 0);
	params.d_rounded = round_multiple(D, 32);
	params.o_ptr = o->data.u8;
	params.o_row_stride = D * Hq;
	params.o_head_stride = D;
	params.o_batch_stride = R * Hq * D;
	params.b = batch_size;
	params.h = Hq;
	params.h_k = Hk;
	params.h_h_k_ratio = Hq / Hk;
	params.scale_softmax = cmd.info.scaled_dot_product_attention.scale;
	params.scale_softmax_log2 = cmd.info.scaled_dot_product_attention.scale * M_LOG2E;
	params.is_causal = cmd.info.scaled_dot_product_attention.is_causal;
	params.p_dropout = 1;
	params.p_dropout_in_uint8_t = 255;
	params.rp_dropout = 1;
	params.scale_softmax_rp_dropout = params.scale_softmax;
	params.window_size_left = ccv_max(R, C);
	params.window_size_right = params.is_causal ? 0 : ccv_max(R, C);
	params.is_seqlens_k_cumulative = true;
	params.dq_ptr = dq->data.u8;
	params.dk_ptr = dk->data.u8;
	params.dv_ptr = dv->data.u8;
	params.dq_row_stride = D * Hq;
	params.dk_row_stride = D * Hq; // This is not a typo, dk / dv is expanded and we sum it later.
	params.dv_row_stride = D * Hq;
	params.dq_head_stride = D;
	params.dk_head_stride = D;
	params.dv_head_stride = D;
	params.dq_batch_stride = R * Hq * D;
	params.dk_batch_stride = C * Hq * D;
	params.dv_batch_stride = C * Hq * D;
	params.do_ptr = d_o->data.u8;
	params.do_row_stride = D * Hq;
	params.do_head_stride = D;
	params.do_batch_stride = R * Hq * D;
	params.deterministic = cmd.info.scaled_dot_product_attention.deterministic;

	size_t dq_accum_size;
	if (params.deterministic)
	{
		const ccv_nnc_cuda_device_prop_t props = ccv_nnc_gpu_device_props();
		const int nsplits = (props.multi_processor_count + batch_size * Hq - 1) / (batch_size * Hq);
		dq_accum_size = sizeof(float) * nsplits * batch_size * params.seqlen_q_rounded * Hq * params.d_rounded;
		params.dq_accum_split_stride = batch_size * params.seqlen_q_rounded * Hq * params.d_rounded;
	} else {
		dq_accum_size = sizeof(float) * batch_size * params.seqlen_q_rounded * Hq * params.d_rounded;
		params.dq_accum_split_stride = 0;
	}

	params.softmax_lse_ptr = saved_softmax_lse->data.u8;
	if (Hq != Hk)
	{
		unsigned char* const workspace = (unsigned char*)ccv_nnc_stream_context_get_workspace(stream_context, sizeof(float) * batch_size * Hq * params.seqlen_q_rounded + dq_accum_size + sizeof(short) * batch_size * Hq * C * D * 2, CCV_TENSOR_GPU_MEMORY);
		params.dsoftmax_sum = workspace;
		params.dq_accum_ptr = workspace + sizeof(float) * batch_size * Hq * params.seqlen_q_rounded;
		params.dk_ptr = workspace + sizeof(float) * batch_size * Hq * params.seqlen_q_rounded + dq_accum_size;
		params.dv_ptr = workspace + sizeof(float) * batch_size * Hq * params.seqlen_q_rounded + dq_accum_size + sizeof(short) * batch_size * Hq * C * D;
	} else {
		unsigned char* const workspace = (unsigned char*)ccv_nnc_stream_context_get_workspace(stream_context, sizeof(float) * batch_size * Hq * params.seqlen_q_rounded + dq_accum_size, CCV_TENSOR_GPU_MEMORY);
		params.dsoftmax_sum = workspace;
		params.dq_accum_ptr = workspace + sizeof(float) * batch_size * Hq * params.seqlen_q_rounded;
		params.dk_accum_ptr = 0;
		params.dv_accum_ptr = 0;
	}
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (params.deterministic)
		hipMemsetAsync(params.dq_accum_ptr, 0, dq_accum_size, stream);
	run_mha_bwd(params, stream);
	CUDA_ENFORCE(hipGetLastError());
	if (Hq != Hk)
	{
		_ccv_nnc_sum_out<<<CUDA_GET_BLOCKS(batch_size * C * Hk * D), CUDA_NUM_THREADS, 0, stream>>>(batch_size * C, Hk, Hq / Hk, D, (__half*)params.dk_ptr, (__half*)dk->data.f16);
		_ccv_nnc_sum_out<<<CUDA_GET_BLOCKS(batch_size * C * Hk * D), CUDA_NUM_THREADS, 0, stream>>>(batch_size * C, Hk, Hq / Hk, D, (__half*)params.dv_ptr, (__half*)dv->data.f16);
		CUDA_ENFORCE(hipGetLastError());
	}
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_SCALED_DOT_PRODUCT_ATTENTION_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA_SM80
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_scaled_dot_product_attention_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SCALED_DOT_PRODUCT_ATTENTION_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA_SM80
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F | CCV_16F | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_scaled_dot_product_attention_back;
#endif
}
