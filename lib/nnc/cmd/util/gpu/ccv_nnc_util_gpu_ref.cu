#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

static int _ccv_nnc_data_transfer(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	int i;
	for (i = 0; i < ccv_min(input_size, output_size); i++)
	{
		const ccv_nnc_tensor_t* a = inputs[i];
		ccv_nnc_tensor_t* b = outputs[i];
		if (a == b)
			continue;
		assert(CCV_IS_TENSOR_CONTIGUOUS(a));
		assert(CCV_IS_TENSOR_CONTIGUOUS(b));
		assert(ccv_nnc_tensor_count(a->info) == ccv_nnc_tensor_count(b->info));
		if (CCV_GET_DATA_TYPE(a->info.datatype) == CCV_QX)
			{ assert(a->info.datatype == b->info.datatype); }
		else
			{ assert(CCV_GET_DATA_TYPE_SIZE(a->info.datatype) == CCV_GET_DATA_TYPE_SIZE(b->info.datatype)); }
		const size_t size = ccv_nnc_tensor_data_size_without_padding(a->info);
		if (stream_context)
		{
			hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
			if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY)
				CUDA_ENFORCE(hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyHostToDevice, stream));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				CUDA_ENFORCE(hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyDeviceToHost, stream));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				CUDA_ENFORCE(hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyHostToHost, stream));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY) {
				const int device_a = CCV_TENSOR_GET_DEVICE_ID(a->info.type);
				const int device_b = CCV_TENSOR_GET_DEVICE_ID(b->info.type);
				if (device_a == device_b)
					CUDA_ENFORCE(hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyDeviceToDevice, stream));
				else
					CUDA_ENFORCE(hipMemcpyPeerAsync(b->data.u8, device_b, a->data.u8, device_a, size, stream));
			}
		} else {
			if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY)
				CUDA_ENFORCE(hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyHostToDevice));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				CUDA_ENFORCE(hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyDeviceToHost));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				CUDA_ENFORCE(hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyHostToHost));
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY) {
				const int device_a = CCV_TENSOR_GET_DEVICE_ID(a->info.type);
				const int device_b = CCV_TENSOR_GET_DEVICE_ID(b->info.type);
				if (device_a == device_b)
					CUDA_ENFORCE(hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyDeviceToDevice));
				else
					CUDA_ENFORCE(hipMemcpyPeer(b->data.u8, device_b, a->data.u8, device_a, size));
			}
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATA_TRANSFER_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_64F | CCV_32F | CCV_16F | CCV_64S | CCV_32S | CCV_8U | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_CPU_MEMORY | CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_data_transfer;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATA_TRANSFER_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_64F | CCV_32F | CCV_16F | CCV_64S | CCV_32S | CCV_8U | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_CPU_MEMORY | CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_data_transfer;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_data_conversion_kernel(const size_t count, const NUM1* const a, NUM2* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = a[i];
	}
}

static int _ccv_nnc_datatype_conversion(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(output_size <= input_size);
	int i;
	for (i = 0; i < output_size; i++)
	{
		const ccv_nnc_tensor_view_t* a = (ccv_nnc_tensor_view_t*)inputs[i];
		ccv_nnc_tensor_view_t* b = (ccv_nnc_tensor_view_t*)outputs[i];
		assert(a != b); // Cannot do inplace transform.
		assert(a->info.format == b->info.format);
		assert(CCV_TENSOR_GET_DEVICE_ID(a->info.type) == CCV_TENSOR_GET_DEVICE_ID(b->info.type));
		hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
		const size_t tensor_count = ccv_nnc_tensor_count(a->info);
		if (a->info.datatype == b->info.datatype) {
			// If it is the same, just do a normal data transfer.
			const size_t size = tensor_count * CCV_GET_DATA_TYPE_SIZE(a->type);
			hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyDeviceToDevice, stream);
		} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, a->data.f32, (__half*)b->data.f16);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			const int tensor_count = ccv_nnc_tensor_count(a->info);
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, (__half*)a->data.f16, b->data.f32);
		} else if (a->info.datatype == CCV_64F && b->info.datatype == CCV_32F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, a->data.f64, b->data.f32);
		} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_64F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			const int tensor_count = ccv_nnc_tensor_count(a->info);
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, a->data.f32, b->data.f64);
		} else if (a->info.datatype == CCV_64F && b->info.datatype == CCV_16F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, a->data.f64, (__half*)b->data.f16);
		} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_64F) {
			assert(CCV_IS_TENSOR_CONTIGUOUS(a));
			assert(CCV_IS_TENSOR_CONTIGUOUS(b));
			const int tensor_count = ccv_nnc_tensor_count(a->info);
			assert(tensor_count == ccv_nnc_tensor_count(b->info));
			_ccv_nnc_data_conversion_kernel<<<CUDA_GET_BLOCKS(tensor_count), CUDA_NUM_THREADS, 0, stream>>>(tensor_count, (__half*)a->data.f16, b->data.f64);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATATYPE_CONVERSION_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_64F | CCV_32F | CCV_16F | CCV_16BF;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_datatype_conversion;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATATYPE_CONVERSION_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_64F | CCV_32F | CCV_16F | CCV_16BF;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_datatype_conversion;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_masked_fill_kernel(const size_t a_count, const size_t b_count, const NUM2 p, const NUM1 q, const NUM1* const a, const NUM2* const b, NUM1* const c)
{
	CUDA_1D_KERNEL_LOOP(i, a_count) {
		c[i] = (b[i % b_count] == p) ? q : a[i];
	}
}

static void _ccv_nnc_masked_fill_gpu_ref(const float p, const float q, ccv_nnc_tensor_view_t* const a, ccv_nnc_tensor_view_t* const b, ccv_nnc_tensor_view_t* const c, ccv_nnc_stream_context_t* const stream_context)
{
	assert(c->info.datatype == a->info.datatype);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const size_t a_count = ccv_nnc_tensor_count(a->info);
	const size_t b_count = ccv_nnc_tensor_count(b->info);
	assert(a_count >= b_count);
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(CCV_IS_TENSOR_CONTIGUOUS(c));
	if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, p, q, a->data.f32, b->data.f32, c->data.f32);
	else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32S)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (int)(p + 0.5), q, a->data.f32, b->data.i32, c->data.f32);
	else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (__half)p, q, a->data.f32, (__half*)b->data.f16, c->data.f32);
	else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, p, (__half)q, (__half*)a->data.f16, b->data.f32, (__half*)c->data.f16);
	else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32S)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (int)(p + 0.5), (__half)q, (__half*)a->data.f16, b->data.i32, (__half*)c->data.f16);
	else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_16F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (__half)p, (__half)q, (__half*)a->data.f16, (__half*)b->data.f16, (__half*)c->data.f16);
	else if (a->info.datatype == CCV_32S && b->info.datatype == CCV_32F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, p, (int)(q + 0.5), a->data.i32, b->data.f32, c->data.i32);
	else if (a->info.datatype == CCV_32S && b->info.datatype == CCV_32S)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (int)(p + 0.5), (int)(q + 0.5), a->data.i32, b->data.i32, c->data.i32);
	else if (a->info.datatype == CCV_32S && b->info.datatype == CCV_16F)
		_ccv_nnc_masked_fill_kernel<<<CUDA_GET_BLOCKS(a_count), CUDA_NUM_THREADS, 0, stream>>>(a_count, b_count, (__half)p, (int)(q + 0.5), a->data.i32, (__half*)b->data.f16, c->data.i32);
}

static int _ccv_nnc_masked_fill_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 2);
	assert(inputs[0]);
	assert(inputs[1]);
	assert(outputs[0]);
	_ccv_nnc_masked_fill_gpu_ref(cmd.info.blas.a[0], cmd.info.blas.a[1], (ccv_nnc_tensor_view_t*)inputs[0], (ccv_nnc_tensor_view_t*)inputs[1], (ccv_nnc_tensor_view_t*)outputs[0], stream_context);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_masked_fill_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 3);
	_ccv_nnc_masked_fill_gpu_ref(cmd.info.blas.a[0], 0, (ccv_nnc_tensor_view_t*)inputs[0], (ccv_nnc_tensor_view_t*)inputs[2], (ccv_nnc_tensor_view_t*)outputs[0], stream_context);
	// TODO: doesn't really support taking gradient on mask.
	// if (output_size >= 2 && outputs[1])
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_MASKED_FILL_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_masked_fill_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_MASKED_FILL_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_masked_fill_back;
}
